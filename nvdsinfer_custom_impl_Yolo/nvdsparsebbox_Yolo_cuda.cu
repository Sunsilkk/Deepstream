#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 *
 * Edited by Marcos Luciano
 * https://www.github.com/marcoslucianops
 */

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "nvdsinfer_custom_impl.h"

#include "utils.h"
#include "yoloPlugins.h"

__global__ void decodeTensor_YOLO_ONNX(NvDsInferParseObjectInfo *binfo, const float* detections, const int numClasses,
    const int outputSize, float netW, float netH, const float* preclusterThreshold, int* numDetections)
{
    uint x_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (x_id >= outputSize)
      return;

    float maxProb = 0.0f;
    int maxIndex = -1;

    for (uint i = 0; i < numClasses; ++i) {
      float prob = detections[x_id * (5 + numClasses) + 5 + i];
      if (prob > maxProb) {
        maxProb = prob;
        maxIndex = i;
      }
    }

    const float objectness = detections[x_id * (5 + numClasses) + 4];

    if (objectness * maxProb < preclusterThreshold[maxIndex])
      return;

    int count = (int)atomicAdd(numDetections, 1);

    const float bxc = detections[x_id * (5 + numClasses) + 0];
    const float byc = detections[x_id * (5 + numClasses) + 1];
    const float bw = detections[x_id * (5 + numClasses) + 2];
    const float bh = detections[x_id * (5 + numClasses) + 3];

    float x0 = bxc - bw / 2;
    float y0 = byc - bh / 2;
    float x1 = x0 + bw;
    float y1 = y0 + bh;
    x0 = fminf(float(netW), fmaxf(float(0.0), x0));
    y0 = fminf(float(netH), fmaxf(float(0.0), y0));
    x1 = fminf(float(netW), fmaxf(float(0.0), x1));
    y1 = fminf(float(netH), fmaxf(float(0.0), y1));

    binfo[count].left = x0;
    binfo[count].top = y0;
    binfo[count].width = fminf(float(netW), fmaxf(float(0.0), x1 - x0));
    binfo[count].height = fminf(float(netH), fmaxf(float(0.0), y1 - y0));
    binfo[count].detectionConfidence = objectness * maxProb;
    binfo[count].classId = maxIndex;
}

__global__ void decodeTensor_YOLOV8_ONNX(NvDsInferParseObjectInfo* binfo, const float* detections, const int numClasses,
    const int outputSize, float netW, float netH, const float* preclusterThreshold, int* numDetections)
{
    uint x_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (x_id >= outputSize)
      return;

    float maxProb = 0.0f;
    int maxIndex = -1;

    for (uint i = 0; i < numClasses; ++i) {
      float prob = detections[x_id + outputSize * (i + 4)];
      if (prob > maxProb) {
        maxProb = prob;
        maxIndex = i;
      }
    }

    if (maxProb < preclusterThreshold[maxIndex])
      return;

    int count = (int)atomicAdd(numDetections, 1);

    const float bxc = detections[x_id + outputSize * 0];
    const float byc = detections[x_id + outputSize * 1];
    const float bw = detections[x_id + outputSize * 2];
    const float bh = detections[x_id + outputSize * 3];

    float x0 = bxc - bw / 2;
    float y0 = byc - bh / 2;
    float x1 = x0 + bw;
    float y1 = y0 + bh;
    x0 = fminf(float(netW), fmaxf(float(0.0), x0));
    y0 = fminf(float(netH), fmaxf(float(0.0), y0));
    x1 = fminf(float(netW), fmaxf(float(0.0), x1));
    y1 = fminf(float(netH), fmaxf(float(0.0), y1));

    binfo[count].left = x0;
    binfo[count].top = y0;
    binfo[count].width = fminf(float(netW), fmaxf(float(0.0), x1 - x0));
    binfo[count].height = fminf(float(netH), fmaxf(float(0.0), y1 - y0));
    binfo[count].detectionConfidence = maxProb;
    binfo[count].classId = maxIndex;
}

__global__ void decodeTensor_YOLOX_ONNX(NvDsInferParseObjectInfo *binfo, const float* detections, const int numClasses,
    const int outputSize, float netW, float netH, const int *grid0, const int *grid1, const int *strides,
    const float* preclusterThreshold, int* numDetections)
{
    uint x_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (x_id >= outputSize)
      return;

    float maxProb = 0.0f;
    int maxIndex = -1;

    for (uint i = 0; i < numClasses; ++i) {
      float prob = detections[x_id * (5 + numClasses) + 5 + i];
      if (prob > maxProb) {
        maxProb = prob;
        maxIndex = i;
      }
    }

    const float objectness = detections[x_id * (5 + numClasses) + 4];

    if (objectness * maxProb < preclusterThreshold[maxIndex])
      return;

    int count = (int)atomicAdd(numDetections, 1);

    const float bxc = (detections[x_id * (5 + numClasses) + 0] + grid0[x_id]) * strides[x_id];
    const float byc = (detections[x_id * (5 + numClasses) + 1] + grid1[x_id]) * strides[x_id];
    const float bw = __expf(detections[x_id * (5 + numClasses) + 2]) * strides[x_id];
    const float bh = __expf(detections[x_id * (5 + numClasses) + 3]) * strides[x_id];

    float x0 = bxc - bw / 2;
    float y0 = byc - bh / 2;
    float x1 = x0 + bw;
    float y1 = y0 + bh;
    x0 = fminf(float(netW), fmaxf(float(0.0), x0));
    y0 = fminf(float(netH), fmaxf(float(0.0), y0));
    x1 = fminf(float(netW), fmaxf(float(0.0), x1));
    y1 = fminf(float(netH), fmaxf(float(0.0), y1));

    binfo[count].left = x0;
    binfo[count].top = y0;
    binfo[count].width = fminf(float(netW), fmaxf(float(0.0), x1 - x0));
    binfo[count].height = fminf(float(netH), fmaxf(float(0.0), y1 - y0));
    binfo[count].detectionConfidence = objectness * maxProb;
    binfo[count].classId = maxIndex;
}

__global__ void decodeTensor_YOLO_NAS_ONNX(NvDsInferParseObjectInfo *binfo, const float* scores, const float* boxes,
    const int numClasses, const int outputSize, float netW, float netH, const float* preclusterThreshold, int* numDetections)
{
    uint x_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (x_id >= outputSize)
      return;

    float maxProb = 0.0f;
    int maxIndex = -1;

    for (uint i = 0; i < numClasses; ++i) {
      float prob = scores[x_id * numClasses + i];
      if (prob > maxProb) {
        maxProb = prob;
        maxIndex = i;
      }
    }

    if (maxProb < preclusterThreshold[maxIndex])
      return;

    int count = (int)atomicAdd(numDetections, 1);

    float x0 = boxes[x_id * 4 + 0];
    float y0 = boxes[x_id * 4 + 1];
    float x1 = boxes[x_id * 4 + 2];
    float y1 = boxes[x_id * 4 + 3];

    x0 = fminf(float(netW), fmaxf(float(0.0), x0));
    y0 = fminf(float(netH), fmaxf(float(0.0), y0));
    x1 = fminf(float(netW), fmaxf(float(0.0), x1));
    y1 = fminf(float(netH), fmaxf(float(0.0), y1));

    binfo[count].left = x0;
    binfo[count].top = y0;
    binfo[count].width = fminf(float(netW), fmaxf(float(0.0), x1 - x0));
    binfo[count].height = fminf(float(netH), fmaxf(float(0.0), y1 - y0));
    binfo[count].detectionConfidence = maxProb;
    binfo[count].classId = maxIndex;
}

__global__ void decodeTensor_PPYOLOE_ONNX(NvDsInferParseObjectInfo *binfo, const float* scores, const float* boxes,
    const int numClasses, const int outputSize, float netW, float netH, const float* preclusterThreshold, int* numDetections)
{
    uint x_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (x_id >= outputSize)
      return;

    float maxProb = 0.0f;
    int maxIndex = -1;

    for (uint i = 0; i < numClasses; ++i) {
      float prob = scores[x_id + outputSize * i];
      if (prob > maxProb) {
        maxProb = prob;
        maxIndex = i;
      }
    }

    if (maxProb < preclusterThreshold[maxIndex])
      return;

    int count = (int)atomicAdd(numDetections, 1);

    float x0 = boxes[x_id * 4 + 0];
    float y0 = boxes[x_id * 4 + 1];
    float x1 = boxes[x_id * 4 + 2];
    float y1 = boxes[x_id * 4 + 3];

    x0 = fminf(float(netW), fmaxf(float(0.0), x0));
    y0 = fminf(float(netH), fmaxf(float(0.0), y0));
    x1 = fminf(float(netW), fmaxf(float(0.0), x1));
    y1 = fminf(float(netH), fmaxf(float(0.0), y1));

    binfo[count].left = x0;
    binfo[count].top = y0;
    binfo[count].width = fminf(float(netW), fmaxf(float(0.0), x1 - x0));
    binfo[count].height = fminf(float(netH), fmaxf(float(0.0), y1 - y0));
    binfo[count].detectionConfidence = maxProb;
    binfo[count].classId = maxIndex;
}

static bool
NvDsInferParseCustom_YOLO_ONNX(std::vector<NvDsInferLayerInfo> const& outputLayersInfo,
    NvDsInferNetworkInfo const& networkInfo, NvDsInferParseDetectionParams const& detectionParams,
    std::vector<NvDsInferParseObjectInfo>& objectList)
{
  if (outputLayersInfo.empty()) {
    std::cerr << "ERROR: Could not find output layer in bbox parsing" << std::endl;
    return false;
  }

  const NvDsInferLayerInfo& layer = outputLayersInfo[0];

  const uint outputSize = layer.inferDims.d[0];
  const uint numClasses = layer.inferDims.d[1] - 5;

  if (numClasses != detectionParams.numClassesConfigured) {
    std::cerr << "WARNING: Number of classes mismatch, make sure to set num-detected-classes=" << numClasses
        << " in config_infer file\n" << std::endl;
  }

  thrust::device_vector<NvDsInferParseObjectInfo> objects(outputSize);

  std::vector<int> numDetections = { 0 };
  thrust::device_vector<int> d_numDetections(numDetections);

  thrust::device_vector<float> preclusterThreshold(detectionParams.perClassPreclusterThreshold);

  int threads_per_block = 1024;
  int number_of_blocks = ((outputSize - 1) / threads_per_block) + 1;

  decodeTensor_YOLO_ONNX<<<threads_per_block, number_of_blocks>>>(
      thrust::raw_pointer_cast(objects.data()), (const float*) (layer.buffer), numClasses, outputSize,
      static_cast<float>(networkInfo.width), static_cast<float>(networkInfo.height),
      thrust::raw_pointer_cast(preclusterThreshold.data()), thrust::raw_pointer_cast(d_numDetections.data()));

  thrust::copy(d_numDetections.begin(), d_numDetections.end(), numDetections.begin());
  objectList.resize(numDetections[0]);
  thrust::copy(objects.begin(), objects.begin() + numDetections[0], objectList.begin());

  return true;
}

static bool
NvDsInferParseCustom_YOLOV8_ONNX(std::vector<NvDsInferLayerInfo> const& outputLayersInfo,
    NvDsInferNetworkInfo const& networkInfo, NvDsInferParseDetectionParams const& detectionParams,
    std::vector<NvDsInferParseObjectInfo>& objectList)
{
  if (outputLayersInfo.empty()) {
    std::cerr << "ERROR: Could not find output layer in bbox parsing" << std::endl;
    return false;
  }

  const NvDsInferLayerInfo& layer = outputLayersInfo[0];

  const uint numClasses = layer.inferDims.d[0] - 4;
  const uint outputSize = layer.inferDims.d[1];

  if (numClasses != detectionParams.numClassesConfigured) {
    std::cerr << "WARNING: Number of classes mismatch, make sure to set num-detected-classes=" << numClasses
        << " in config_infer file\n" << std::endl;
  }

  thrust::device_vector<NvDsInferParseObjectInfo> objects(outputSize);

  std::vector<int> numDetections = { 0 };
  thrust::device_vector<int> d_numDetections(numDetections);

  thrust::device_vector<float> preclusterThreshold(detectionParams.perClassPreclusterThreshold);

  int threads_per_block = 1024;
  int number_of_blocks = ((outputSize - 1) / threads_per_block) + 1;

  decodeTensor_YOLOV8_ONNX<<<threads_per_block, number_of_blocks>>>(
      thrust::raw_pointer_cast(objects.data()), (const float*) (layer.buffer), numClasses, outputSize,
      static_cast<float>(networkInfo.width), static_cast<float>(networkInfo.height),
      thrust::raw_pointer_cast(preclusterThreshold.data()), thrust::raw_pointer_cast(d_numDetections.data()));

  thrust::copy(d_numDetections.begin(), d_numDetections.end(), numDetections.begin());
  objectList.resize(numDetections[0]);
  thrust::copy(objects.begin(), objects.begin() + numDetections[0], objectList.begin());

  return true;
}

static bool
NvDsInferParseCustom_YOLOX_ONNX(std::vector<NvDsInferLayerInfo> const& outputLayersInfo,
    NvDsInferNetworkInfo const& networkInfo, NvDsInferParseDetectionParams const& detectionParams,
    std::vector<NvDsInferParseObjectInfo>& objectList)
{
  if (outputLayersInfo.empty()) {
    std::cerr << "ERROR: Could not find output layer in bbox parsing" << std::endl;
    return false;
  }

  const NvDsInferLayerInfo& layer = outputLayersInfo[0];

  const uint outputSize = layer.inferDims.d[0];
  const uint numClasses = layer.inferDims.d[1] - 5;

  if (numClasses != detectionParams.numClassesConfigured) {
    std::cerr << "WARNING: Number of classes mismatch, make sure to set num-detected-classes=" << numClasses
        << " in config_infer file\n" << std::endl;
  }

  thrust::device_vector<NvDsInferParseObjectInfo> objects(outputSize);

  std::vector<int> numDetections = { 0 };
  thrust::device_vector<int> d_numDetections(numDetections);

  thrust::device_vector<float> preclusterThreshold(detectionParams.perClassPreclusterThreshold);

  std::vector<int> strides = {8, 16, 32};

  std::vector<int> grid0;
  std::vector<int> grid1;
  std::vector<int> gridStrides;

  for (uint s = 0; s < strides.size(); ++s) {
    int num_grid_y = networkInfo.height / strides[s];
    int num_grid_x = networkInfo.width / strides[s];
    for (int g1 = 0; g1 < num_grid_y; ++g1) {
      for (int g0 = 0; g0 < num_grid_x; ++g0) {
        grid0.push_back(g0);
        grid1.push_back(g1);
        gridStrides.push_back(strides[s]);
      }
    }
  }

  thrust::device_vector<int> d_grid0(grid0);
  thrust::device_vector<int> d_grid1(grid1);
  thrust::device_vector<int> d_gridStrides(gridStrides);

  int threads_per_block = 1024;
  int number_of_blocks = ((outputSize - 1) / threads_per_block) + 1;

  decodeTensor_YOLOX_ONNX<<<threads_per_block, number_of_blocks>>>(
      thrust::raw_pointer_cast(objects.data()), (const float*) (layer.buffer), numClasses, outputSize,
      static_cast<float>(networkInfo.width), static_cast<float>(networkInfo.height),
      thrust::raw_pointer_cast(d_grid0.data()), thrust::raw_pointer_cast(d_grid1.data()),
      thrust::raw_pointer_cast(d_gridStrides.data()), thrust::raw_pointer_cast(preclusterThreshold.data()),
      thrust::raw_pointer_cast(d_numDetections.data()));

  thrust::copy(d_numDetections.begin(), d_numDetections.end(), numDetections.begin());
  objectList.resize(numDetections[0]);
  thrust::copy(objects.begin(), objects.begin() + numDetections[0], objectList.begin());

  return true;
}

static bool
NvDsInferParseCustom_YOLO_NAS_ONNX(std::vector<NvDsInferLayerInfo> const& outputLayersInfo,
    NvDsInferNetworkInfo const& networkInfo, NvDsInferParseDetectionParams const& detectionParams,
    std::vector<NvDsInferParseObjectInfo>& objectList)
{
  if (outputLayersInfo.empty()) {
    std::cerr << "ERROR: Could not find output layer in bbox parsing" << std::endl;
    return false;
  }

  const NvDsInferLayerInfo& scores = outputLayersInfo[0];
  const NvDsInferLayerInfo& boxes = outputLayersInfo[1];

  const uint outputSize = scores.inferDims.d[0];
  const uint numClasses = scores.inferDims.d[1];

  if (numClasses != detectionParams.numClassesConfigured) {
    std::cerr << "WARNING: Number of classes mismatch, make sure to set num-detected-classes=" << numClasses
        << " in config_infer file\n" << std::endl;
  }

  thrust::device_vector<NvDsInferParseObjectInfo> objects(outputSize);

  std::vector<int> numDetections = { 0 };
  thrust::device_vector<int> d_numDetections(numDetections);

  thrust::device_vector<float> preclusterThreshold(detectionParams.perClassPreclusterThreshold);

  int threads_per_block = 1024;
  int number_of_blocks = ((outputSize - 1) / threads_per_block) + 1;

  decodeTensor_YOLO_NAS_ONNX<<<threads_per_block, number_of_blocks>>>(
      thrust::raw_pointer_cast(objects.data()), (const float*) (scores.buffer), (const float*) (boxes.buffer), numClasses,
      outputSize, static_cast<float>(networkInfo.width), static_cast<float>(networkInfo.height),
      thrust::raw_pointer_cast(preclusterThreshold.data()), thrust::raw_pointer_cast(d_numDetections.data()));

  thrust::copy(d_numDetections.begin(), d_numDetections.end(), numDetections.begin());
  objectList.resize(numDetections[0]);
  thrust::copy(objects.begin(), objects.begin() + numDetections[0], objectList.begin());

  return true;
}

static bool
NvDsInferParseCustom_PPYOLOE_ONNX(std::vector<NvDsInferLayerInfo> const& outputLayersInfo,
    NvDsInferNetworkInfo const& networkInfo, NvDsInferParseDetectionParams const& detectionParams,
    std::vector<NvDsInferParseObjectInfo>& objectList)
{
  if (outputLayersInfo.empty()) {
    std::cerr << "ERROR: Could not find output layer in bbox parsing" << std::endl;
    return false;
  }

  const NvDsInferLayerInfo& scores = outputLayersInfo[0];
  const NvDsInferLayerInfo& boxes = outputLayersInfo[1];

  const uint numClasses = scores.inferDims.d[0];
  const uint outputSize = scores.inferDims.d[1];

  if (numClasses != detectionParams.numClassesConfigured) {
    std::cerr << "WARNING: Number of classes mismatch, make sure to set num-detected-classes=" << numClasses
        << " in config_infer file\n" << std::endl;
  }

  thrust::device_vector<NvDsInferParseObjectInfo> objects(outputSize);

  std::vector<int> numDetections = { 0 };
  thrust::device_vector<int> d_numDetections(numDetections);

  thrust::device_vector<float> preclusterThreshold(detectionParams.perClassPreclusterThreshold);

  int threads_per_block = 1024;
  int number_of_blocks = ((outputSize - 1) / threads_per_block) + 1;

  decodeTensor_PPYOLOE_ONNX<<<threads_per_block, number_of_blocks>>>(
      thrust::raw_pointer_cast(objects.data()), (const float*) (scores.buffer), (const float*) (boxes.buffer), numClasses,
      outputSize, static_cast<float>(networkInfo.width), static_cast<float>(networkInfo.height),
      thrust::raw_pointer_cast(preclusterThreshold.data()), thrust::raw_pointer_cast(d_numDetections.data()));

  thrust::copy(d_numDetections.begin(), d_numDetections.end(), numDetections.begin());
  objectList.resize(numDetections[0]);
  thrust::copy(objects.begin(), objects.begin() + numDetections[0], objectList.begin());

  return true;
}

extern "C" bool
NvDsInferParse_YOLO_ONNX(std::vector<NvDsInferLayerInfo> const& outputLayersInfo, NvDsInferNetworkInfo const& networkInfo,
    NvDsInferParseDetectionParams const& detectionParams, std::vector<NvDsInferParseObjectInfo>& objectList)
{
  return NvDsInferParseCustom_YOLO_ONNX(outputLayersInfo, networkInfo, detectionParams, objectList);
}

extern "C" bool
NvDsInferParse_YOLOV8_ONNX(std::vector<NvDsInferLayerInfo> const& outputLayersInfo, NvDsInferNetworkInfo const& networkInfo,
    NvDsInferParseDetectionParams const& detectionParams, std::vector<NvDsInferParseObjectInfo>& objectList)
{
  return NvDsInferParseCustom_YOLOV8_ONNX(outputLayersInfo, networkInfo, detectionParams, objectList);
}

extern "C" bool
NvDsInferParse_YOLOX_ONNX(std::vector<NvDsInferLayerInfo> const& outputLayersInfo, NvDsInferNetworkInfo const& networkInfo,
    NvDsInferParseDetectionParams const& detectionParams, std::vector<NvDsInferParseObjectInfo>& objectList)
{
  return NvDsInferParseCustom_YOLOX_ONNX(outputLayersInfo, networkInfo, detectionParams, objectList);
}

extern "C" bool
NvDsInferParse_YOLO_NAS_ONNX(std::vector<NvDsInferLayerInfo> const& outputLayersInfo, NvDsInferNetworkInfo const& networkInfo,
    NvDsInferParseDetectionParams const& detectionParams, std::vector<NvDsInferParseObjectInfo>& objectList)
{
  return NvDsInferParseCustom_YOLO_NAS_ONNX(outputLayersInfo, networkInfo, detectionParams, objectList);
}

extern "C" bool
NvDsInferParse_PPYOLOE_ONNX(std::vector<NvDsInferLayerInfo> const& outputLayersInfo, NvDsInferNetworkInfo const& networkInfo,
    NvDsInferParseDetectionParams const& detectionParams, std::vector<NvDsInferParseObjectInfo>& objectList)
{
  return NvDsInferParseCustom_PPYOLOE_ONNX(outputLayersInfo, networkInfo, detectionParams, objectList);
}
